#include "hip/hip_runtime.h"
#include "softmax_activationnloss.h"

using namespace global;

namespace layer {

	SoftmaxAnL::SoftmaxAnL(Layer* _prev, float* _label, int _class_num, int _batch) : Layer() 
	{
		prev = _prev;
		prev->next = this;

		label = _label; // gpu data
		class_num = _class_num;
		batch = _batch;
		data_size = batch; // �����С
		param_size = 0;
		param_bias_size = 0;

		callCudnn(hipdnnCreateTensorDescriptor(&t_data));
		callCudnn(hipdnnSetTensor4dDescriptor(
			t_data,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			batch,
			class_num,
			1,
			1));

		callCuda(hipMalloc(&tmp_data, sizeof(float) * prev->data_size)); // ���softmax����ĸ���ֵa
		callCuda(hipMalloc(&data, sizeof(float) * 1)); // Loss
		callCuda(hipMalloc(&diff, sizeof(float) * prev->data_size)); // diff 
		callCuda(hipMalloc(&predict_label, sizeof(float) * data_size)); // �˴����ڴ�� predict
	}

	SoftmaxAnL::~SoftmaxAnL() 
	{
		callCudnn(hipdnnDestroyTensorDescriptor(t_data));

		callCuda(hipFree(tmp_data));
		callCuda(hipFree(data));
		callCuda(hipFree(diff));
		callCuda(hipFree(predict_label));
	}


	__global__ void corssEntropyLoss(float *softmax_output_a, float *label, int class_num, int batch, float *predict_label, float *loss)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < batch)
		{
			int label_value = 0;
			float max = -1;
			for (int i = 0; i < class_num; i++) {
				if (softmax_output_a[idx * class_num + i] > max) {
					max = softmax_output_a[idx * class_num + i];
					label_value = i;
				}
			}
			predict_label[idx] = (float)label_value;

			atomicAdd(loss, -log(softmax_output_a[idx * class_num + (int)label[idx]]));
		}
	}

	// ���㽻������ʧ��softmax�������ݣ�δ��һ�����ĵ�����diff = f(zl)-1�������Ҫ�Ƚ�softmax���������ֵ��a������diff������labelλ�ü�1
	__global__ void softmaxDiff(const float *label, int class_num, int batch, float *diff)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < batch)
		{
			const int label_value = static_cast<int>(label[idx]);
			diff[idx * class_num + label_value] -= 1.0f; // ��zi��
		}
	}

	void SoftmaxAnL::forward(bool train)
	{
		float a = 1;
		float b = 0;
		callCudnn(hipdnnSoftmaxForward(
			cudnnHandle,
			HIPDNN_SOFTMAX_FAST,
			HIPDNN_SOFTMAX_MODE_CHANNEL,
			&a,
			t_data,
			prev->data,
			&b,
			t_data,
			tmp_data));

		net_utils::setGpuValue(data, 1, 0); // loss = 0
		corssEntropyLoss <<< (batch + 127) / 128, 128 >>> (tmp_data, label, class_num, batch, predict_label, data);
	}

	void SoftmaxAnL::backward()
	{
		callCuda(hipMemcpy(diff, tmp_data, sizeof(float) * prev->data_size, hipMemcpyDeviceToDevice));
		softmaxDiff <<< (batch + 127) / 128, 128 >>> (label, class_num, batch, diff);
	}

	void SoftmaxAnL::update()
	{
		//#                        .::::.
		//#                      .::::::::.
		//#                     :::::::::::
		//#                  ..:::::::::::'
		//#               '::::::::::::'
		//#                 .::::::::::
		//#            '::::::::::::::..
		//#                 ..::::::::::::.
		//#               ``::::::::::::::::
		//#                ::::``:::::::::'        .:::.
		//#               ::::'   ':::::'       .::::::::.
		//#             .::::'      ::::     .:::::::'::::.
		//#            .:::'       :::::  .:::::::::' ':::::.
		//#           .::'        :::::.:::::::::'      ':::::.
		//#          .::'         ::::::::::::::'         ``::::.
		//#      ...:::           ::::::::::::'              ``::.
		//#     ```` ':.          ':::::::::'                  ::::..
		//#                        '.:::::'                    ':'````..
		//#                     ��Ů���� ����BUG
	}
}
