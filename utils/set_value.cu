#include "hip/hip_runtime.h"
#include "set_value.h"

using namespace global;

namespace net_utils {

__global__ void setValue(float* x, int n, float val) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
		x[i] = val;
}

__global__ void dropout(float* x, float* t, int n, float threshold) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
		if (t[i] < threshold)
			x[i] = 0;
}

__global__ void scale(float* x, int n, float epsilon) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
		x[i] = x[i] * epsilon * 2 - epsilon;
}

void setGpuValue(float* x, int n, float val) {
	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	setValue<<<blocksPerGrid, threadsPerBlock>>>(x, n, val);
}

void setGpuUniformValue(float* x, int n, float epsilon) {
	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));
	hiprandGenerateUniform(generator, x, n);
	scale<<<blocksPerGrid, threadsPerBlock>>>(x, n, epsilon);
	hiprandDestroyGenerator(generator);
}

void setGpuNormalValue(float* x, int n, float mean, float stddev) {
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(generator, 888ULL);
	hiprandGenerateNormal(generator, x, n, mean, stddev);
	hiprandDestroyGenerator(generator);
}

void dropGpuValue(float *x, int n, float dropout_rate) {
	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));
	float* t;
	hipMalloc((void**)&t, sizeof(float) * n);
	hiprandGenerateUniform(generator, t, n);
	dropout<<<blocksPerGrid, threadsPerBlock>>>(x, t, n, dropout_rate);
	hipFree(t);
	hiprandDestroyGenerator(generator);
}

void scaleGpuValue(float *x, int n, float scale) {
	hipblasSscal(cublasHandle, n, &scale, x, 1);
}

}
